#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void squareKernel(int* data, int N);

int main(int argc, char** argv)
{
	int* h_data;
	int* d_data;
	//���������� ���������
	int n = 1000;
	//����� ���������
	int sum = 0;

	// �������� page-locked ������ �� �����
	// ��� ������� ����� ����� ������������ �������� ��� ��������� ������������� �������� ��� ������ ������� ����� ������ � �����������.
	hipHostAlloc(&h_data, n * sizeof(int), hipHostMallocPortable);

	// �������� ������ �� ����������
	hipMalloc(&d_data, n * sizeof(int));
	
	dim3 block(512);
	dim3 grid((n + block.x - 1) / block.x);

	//grid - ���������� ������
	//block - ������ �����
	squareKernel<<<grid, block>>>(d_data, n);

	//�������� ������ � ���������� (d_data) �� ���� (h_data)
	hipMemcpy(h_data, d_data, n * sizeof(int), hipMemcpyDeviceToHost);

	for (int j = 0; j < n; j++)
	{
		sum = sum + h_data[j];
	}

	printf("sum = %d\n", sum);
	return 0;
}

__global__ void squareKernel(int* data, int N)
{
	//threadIdx � ����� ���� � �����
	//blockIdx � ����� �����, � ������� ��������� ����
	//blockDim � ������ �����

	//���������� ������ ���� ������ ����
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
	{
		data[i] = i * i;
	}
}